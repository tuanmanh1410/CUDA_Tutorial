#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

double seconds(){
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp,&tzp);
    return ((double)tp.tv_sec+(double)tp.tv_usec*1.e-6);
    }

void initialData(float *ip, int size){
    for (int i = 0; i < size; i ++){
        ip[i] =  (float)(rand() & 0xFF)/100.0f;
        }
    return;
    }

void checkResult(float *hostRef, float *gpuRef, const int N){
    double epsilon = 1.0e-8;
    for (int i=0; i < N; i++){
        if(abs(hostRef[i]-gpuRef[i])>epsilon){
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n",hostRef[i],gpuRef[i],i);
            break;
            }
        }
    return;
    }

__global__ void readOffset(float *A,float *B,float *C,const int n,int offset){
    // assigned offset value
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;
    if (k<n) C[i] = A[k] + B[k];
    }

void sumArraysOnHost(float *A, float *B, float *C, const int n, int offset){
    for (int idx = offset,k=0; idx < n; idx++, k++){
        C[k] = A[idx] + B[idx];
        }
    }


int main(int argc, char **argv){
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,dev);
    printf("%s starting reduction at ",argv[0]);
    printf("device %d: %s ",dev,deviceProp.name);
    hipSetDevice(dev);

    // set up array size
    int nElem = 1 << 20; // total number of elements to reduce
    printf(" with array size %d \n",nElem);
    size_t nBytes = nElem * sizeof(float);

    // set up offset for summary
    int blocksize = 512;
    int offset = 0;

    if (argc > 1) offset = atoi(argv[1]);
    if (argc > 2) offset = atoi(argv[2]);

    // execution configuration
    dim3 block(blocksize,1);
    dim3 grid((nElem+block.x-1)/block.x,1);

    // allocate host mem
    float *h_A = (float *)malloc(nBytes);
    float *h_B = (float *)malloc(nBytes);
    float *hostRef = (float *)malloc(nBytes);
    float *gpuRef = (float *)malloc(nBytes);

    // initialize host array
    initialData(h_A,nElem);
    memcpy(h_B,h_A,nBytes);

    // summary at host side
    sumArraysOnHost(h_A,h_B,hostRef,nElem,offset);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A,nBytes);
    hipMalloc((float **)&d_B,nBytes);
    hipMalloc((float **)&d_C,nBytes);

    // copy data from host to device
    hipMemcpy(d_A,h_A,nBytes,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_A,nBytes,hipMemcpyHostToDevice);

   
    double iStart = seconds();
    readOffset<<<grid,block>>>(d_A,d_B,d_C,nElem,offset);
    hipDeviceSynchronize();
    double iElaps = seconds() - iStart;
    printf("readoffset <<<%4d,%4d>>> offset %4d elapsed %f sec\n",grid.x,block.x,offset,iElaps);
    hipGetLastError();

    // copy kernel result back to host side and check results
    hipMemcpy(gpuRef,d_C,nBytes,hipMemcpyDeviceToHost);
    checkResult(hostRef,gpuRef,nElem-offset);

    // free host and device mem
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);

    // reset device
    hipDeviceReset();
    return EXIT_SUCCESS;
    }


