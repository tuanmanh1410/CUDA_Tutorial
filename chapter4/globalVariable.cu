#include <hip/hip_runtime.h>
#include <stdio.h>

// define the global variable devData
__device__ float devData;

__global__ void checkGlobalVariable(){
    // display the original value
    printf("Device: the value of the global variable is %f\n",devData);
    // alter the value
    devData += 2.0f;
    }

int main(void){
    // initialize the global variable
    float value = 3.14f;
    /*
    Attention: You cannot use 
        cudaMemcpy(&devData,&value,sizeof(float),cudaMemcpyHostToDevice)
    To copy value from host to device on devData.
    */
    hipMemcpyToSymbol(HIP_SYMBOL(devData),&value,sizeof(float));
    printf("Host: copied %f to the global variable\n",value);

    // invoke the kernel
    checkGlobalVariable<<<1,1>>>();

    // copy the global variable back to the host
    hipMemcpyFromSymbol(&value,HIP_SYMBOL(devData),sizeof(float));
    printf("Host: the value changed by the kernel to %f\n",value);

    hipDeviceReset();
    return EXIT_SUCCESS;
    }

